#include "hip/hip_runtime.h"

#include <hip/hip_runtime_api.h>
#include <math.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hipsparse.h>
#include <hipblas.h>
#include <thrust/device_ptr.h>
#include <thrust/sequence.h>

#define gpuCheck(stmt)                                               \
  do {                                                               \
      hipError_t err = stmt;                                        \
      if (err != hipSuccess) {                                      \
          printf("ERROR. Failed to run stmt %s\n", #stmt);           \
          break;                                                     \
      }                                                              \
  } while (0)

// Macro to check the cuBLAS status
#define cublasCheck(stmt)                                            \
  do {                                                               \
      hipblasStatus_t err = stmt;                                     \
      if (err != HIPBLAS_STATUS_SUCCESS) {                            \
          printf("ERROR. Failed to run cuBLAS stmt %s\n", #stmt);    \
          break;                                                     \
      }                                                              \
  } while (0)

// Macro to check the cuSPARSE status
#define cusparseCheck(stmt)                                          \
  do {                                                               \
      hipsparseStatus_t err = stmt;                                   \
      if (err != HIPSPARSE_STATUS_SUCCESS) {                          \
          printf("ERROR. Failed to run cuSPARSE stmt %s\n", #stmt);  \
          break;                                                     \
      }                                                              \
  } while (0)


struct timeval t_start, t_end;
void cputimer_start(){
  gettimeofday(&t_start, 0);
}
void cputimer_stop(const char* info){
  gettimeofday(&t_end, 0);
  double time = (1000000.0*(t_end.tv_sec-t_start.tv_sec) + t_end.tv_usec-t_start.tv_usec);
  printf("Timing - %s. \t\tElasped %.0f microseconds \n", info, time);
}

// Initialize the sparse matrix needed for the heat time step
void matrixInit(double* A, int* ArowPtr, int* AcolIndx, int dimX,
    double alpha) {
  // Stencil from the finete difference discretization of the equation
  double stencil[] = { 1, -2, 1 };
  // Variable holding the position to insert a new element
  size_t ptr = 0;
  // Insert a row of zeros at the beginning of the matrix
  ArowPtr[1] = ptr;
  // Fill the non zero entries of the matrix
  for (int i = 1; i < (dimX - 1); ++i) {
    // Insert the elements: A[i][i-1], A[i][i], A[i][i+1]
    for (int k = 0; k < 3; ++k) {
      // Set the value for A[i][i+k-1]
      A[ptr] = stencil[k];
      // Set the column index for A[i][i+k-1]
      AcolIndx[ptr++] = i + k - 1;
    }
    // Set the number of newly added elements
    ArowPtr[i + 1] = ptr;
  }
  // Insert a row of zeros at the end of the matrix
  ArowPtr[dimX] = ptr;
}

int main(int argc, char **argv) {
  int device = 0;            // Device to be used
  int dimX;                  // Dimension of the metal rod
  int nsteps;                // Number of time steps to perform
  double alpha = 0.4;        // Diffusion coefficient
  double* temp;              // Array to store the final time step
  double* A;                 // Sparse matrix A values in the CSR format
  int* ARowPtr;              // Sparse matrix A row pointers in the CSR format
  int* AColIndx;             // Sparse matrix A col values in the CSR format
  int nzv;                   // Number of non zero values in the sparse matrix
  double* tmp;               // Temporal array of dimX for computations
  size_t bufferSize = 0;     // Buffer size needed by some routines
  void* buffer = nullptr;    // Buffer used by some routines in the libraries
  int concurrentAccessQ ;     // Check if concurrent access flag is set
  double zero = 0;           // Zero constant
  double one = 1;            // One constant
  double norm;               // Variable for norm values
  double error;              // Variable for storing the relative error
  double tempLeft = 200.;    // Left heat source applied to the rod
  double tempRight = 300.;   // Right heat source applied to the rod
  hipblasHandle_t cublasHandle;      // cuBLAS handle
  hipsparseHandle_t cusparseHandle;  // cuSPARSE handle
  hipsparseSpMatDescr_t Adescriptor;   // Mat descriptor needed by cuSPARSE

  // Read the arguments from the command line
  dimX = atoi(argv[1]);
  nsteps = atoi(argv[2]);

  // Print input arguments
  printf("The X dimension of the grid is %d \n", dimX);
  printf("The number of time steps to perform is %d \n", nsteps);

  // Get if the hipDeviceAttributeConcurrentManagedAccess flag is set
  gpuCheck(hipDeviceGetAttribute(&concurrentAccessQ, hipDeviceAttributeConcurrentManagedAccess, device));

  // Calculate the number of non zero values in the sparse matrix. This number
  // is known from the structure of the sparse matrix
  nzv = 3 * dimX - 6;

  //@@ Insert the code to allocate the temp, tmp and the sparse matrix
  //@@ arrays using Unified Memory
  cputimer_start();

  gpuCheck(hipMallocManaged(&temp, sizeof(double) * dimX));
  gpuCheck(hipMallocManaged(&tmp, sizeof(double) * dimX));
  gpuCheck(hipMallocManaged(&A, sizeof(double) * dimX * dimX));
  gpuCheck(hipMallocManaged(&ARowPtr, sizeof(int) * dimX));
  gpuCheck(hipMallocManaged(&AColIndx, sizeof(int) * dimX));

  cputimer_stop("Allocating device memory");

  concurrentAccessQ = 1;

  // Check if concurrentAccessQ is non zero in order to prefetch memory
  if (concurrentAccessQ) {
    cputimer_start();
    //@@ Insert code to prefetch in Unified Memory asynchronously to CPU
    gpuCheck(hipMemPrefetchAsync(temp, sizeof(double)*dimX, hipCpuDeviceId,NULL));
    gpuCheck(hipMemPrefetchAsync(tmp, sizeof(double)*dimX, hipCpuDeviceId, NULL));
    gpuCheck(hipMemPrefetchAsync(A, sizeof(double) * dimX , hipCpuDeviceId, NULL));
    gpuCheck(hipMemPrefetchAsync(ARowPtr, sizeof(int) * dimX, hipCpuDeviceId, NULL));
    gpuCheck(hipMemPrefetchAsync(AColIndx, sizeof(int) * dimX, hipCpuDeviceId, NULL));


    cputimer_stop("Prefetching GPU memory to the host");
  }

  // Initialize the sparse matrix
  cputimer_start();
  matrixInit(A, ARowPtr, AColIndx, dimX, alpha);
  cputimer_stop("Initializing the sparse matrix on the host");

  //Initiliaze the boundary conditions for the heat equation
  cputimer_start();
  memset(temp, 0, sizeof(double) * dimX);
  temp[0] = tempLeft;
  temp[dimX - 1] = tempRight;
  cputimer_stop("Initializing memory on the host");

  


  if (concurrentAccessQ) {
    cputimer_start();
    //@@ Insert code to prefetch in Unified Memory asynchronously to the GPU
    
    gpuCheck(hipMemPrefetchAsync(temp, sizeof(double) * dimX, device, NULL));
    gpuCheck(hipMemPrefetchAsync(tmp, sizeof(double) * dimX, device, NULL));
    gpuCheck(hipMemPrefetchAsync(A, sizeof(double) * dimX, device, NULL));
    gpuCheck(hipMemPrefetchAsync(ARowPtr, sizeof(int) * dimX, device, NULL));
    gpuCheck(hipMemPrefetchAsync(AColIndx, sizeof(int) * dimX, device, NULL));

    
    cputimer_stop("Prefetching GPU memory to the device");
  }

  //@@ Insert code to create the cuBLAS handle
  cublasCheck(hipblasCreate(&cublasHandle));

  //@@ Insert code to create the cuSPARSE handle
  cusparseCheck(hipsparseCreate(&cusparseHandle));
  
  //@@ Insert code to set the cuBLAS pointer mode to HIPSPARSE_POINTER_MODE_HOST
  cusparseCheck(hipsparseSetPointerMode(cusparseHandle,HIPSPARSE_POINTER_MODE_HOST));
   

  //@@ Insert code to call cusparse api to create the mat descriptor used by cuSPARSE
  Adescriptor = NULL;
  cusparseCheck(hipsparseCreateMatDescr((hipsparseMatDescr_t*)&Adescriptor));
  cusparseCheck(hipsparseSetMatType((hipsparseMatDescr_t)Adescriptor, HIPSPARSE_MATRIX_TYPE_GENERAL));
  cusparseCheck(hipsparseSetMatIndexBase((hipsparseMatDescr_t)Adescriptor, HIPSPARSE_INDEX_BASE_ZERO));

 
  //@@ Insert code to call cusparse api to get the buffer size needed by the sparse matrix per
  //@@ vector (SMPV) CSR routine of cuSPARSE

  cusparseCheck(hipsparseCreateCsr(&Adescriptor, dimX, dimX, nzv,ARowPtr, AColIndx, A,HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F));
  hipsparseDnVecDescr_t vecx = NULL;
  cusparseCheck(hipsparseCreateDnVec(&vecx, dimX, temp, HIP_R_32F));
  hipsparseDnVecDescr_t vecAx = NULL;
  cusparseCheck(hipsparseCreateDnVec(&vecAx, dimX, tmp, HIP_R_32F));

  //cusparseCheck(hipsparseSpMV_bufferSize(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,1, A, temp, 0, tmp, HIP_R_32F,HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize))


  //@@ Insert code to allocate the buffer needed by cuSPARSE

  cusparseCheck(hipsparseSpMV_bufferSize(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, Adescriptor, vecx, &one, vecAx, HIP_R_32F, HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize));
  //cusparseCheck(hipMalloc(&buffer, bufferSize));




  // Perform the time step iterations
  for (int it = 0; it < nsteps; ++it) {
    //@@ Insert code to call cusparse api to compute the SMPV (sparse matrix multiplication) for
    //@@ the CSR matrix using cuSPARSE. This calculation corresponds to:
    //@@ tmp = 1 * A * temp + 0 * tmp
      
      if (it == 0)
          cputimer_start();
            cusparseCheck(hipsparseSpMV(cusparseHandle,HIPSPARSE_OPERATION_NON_TRANSPOSE,&one,Adescriptor,vecx,&zero,vecAx,HIP_R_32F,HIPSPARSE_SPMV_ALG_DEFAULT,&bufferSize));
    
    //cusparseCheck(hipsparseSpMV(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,1, A, temp, 0, tmp, HIP_R_32F,HIPSPARSE_SPMV_ALG_DEFAULT, buffer));

    //@@ Insert code to call cublas api to compute the axpy routine using cuBLAS.
    //@@ This calculation corresponds to: temp = alpha * tmp + temp

    cublasCheck(hipblasDaxpy(cublasHandle, dimX, &alpha, tmp, one, temp, one));
    if (it == 0)
        cputimer_stop("SpMV time:");

    

    //@@ Insert code to call cublas api to compute the norm of the vector using cuBLAS
    //@@ This calculation corresponds to: ||temp||

    cublasCheck(hipblasDnrm2(cublasHandle, dimX, temp,one, &norm));


    // If the norm of A*temp is smaller than 10^-4 exit the loop
    if (norm < 1e-4)
      break;
  }

  // Calculate the exact solution using thrust
  thrust::device_ptr<double> thrustPtr(tmp);
  thrust::sequence(thrustPtr, thrustPtr + dimX, tempLeft,
      (tempRight - tempLeft) / (dimX - 1));

  // Calculate the relative approximation error:
  one = -1;
  //@@ Insert the code to call cublas api to compute the difference between the exact solution
  //@@ and the approximation
  //@@ This calculation corresponds to: tmp = -temp + tmp

  cublasCheck(hipblasDaxpy(cublasHandle, dimX, &one, temp, 1, tmp, 1));

  //@@ Insert the code to call cublas api to compute the norm of the absolute error
  //@@ This calculation corresponds to: || tmp ||

  cublasCheck(hipblasDnrm2(cublasHandle, dimX, tmp, 1, &norm)); 

  error = norm;
  //@@ Insert the code to call cublas api to compute the norm of temp
  //@@ This calculation corresponds to: || temp ||

  cublasCheck(hipblasDnrm2(cublasHandle, dimX, temp, 1, &norm));
  
  // Calculate the relative error
  error = error / norm;
  printf("The relative error of the approximation is %f\n", error);

  // FLOPS 
  //float flops = nzv
  printf("FLOP %d\n", nzv);

  //@@ Insert the code to destroy the mat descriptor
  
  cusparseCheck(hipsparseDestroySpMat(Adescriptor));

  //@@ Insert the code to destroy the cuSPARSE handle

  cusparseCheck(hipsparseDestroy(cusparseHandle));

  //@@ Insert the code to destroy the cuBLAS handle
  cublasCheck(hipblasDestroy(cublasHandle));

  cusparseCheck(hipsparseDestroyDnVec(vecx));
  cusparseCheck(hipsparseDestroyDnVec(vecAx));

  //@@ Insert the code for deallocating memory

  free(buffer);
  gpuCheck(hipFree(temp));
  gpuCheck(hipFree(tmp));
  gpuCheck(hipFree(A));
  gpuCheck(hipFree(ARowPtr));
  gpuCheck(hipFree(AColIndx));

  return 0;
}
